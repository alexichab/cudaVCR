#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "mc_growth.h"

// Генерация случайных смещений на GPU
__device__ void random_displacements_gpu(float* ax_, float* ay_, float* az_, unsigned short int config_, float* transform_array, hiprandState* state, float T) {
    if (config_ >= Nconfig || config_ == 65535) {
        *ax_ = 0.0f; *ay_ = 0.0f; *az_ = 0.0f;
        return;
    }
    if (T <= 0) {
        *ax_ = 0.0f; *ay_ = 0.0f; *az_ = 0.0f;
        return;
    }
    float a1 = hiprand_normal(state);
    float a2 = hiprand_normal(state);
    float a3 = hiprand_normal(state);
    float coeff = sqrtf(0.5f * T);
    float* p = transform_array + config_ * 6;
    *ax_ = coeff * (p[0] * a1 + p[3] * a2 + p[4] * a3);
    *ay_ = coeff * (p[3] * a1 + p[1] * a2 + p[5] * a3);
    *az_ = coeff * (p[4] * a1 + p[5] * a2 + p[2] * a3);
}

// Реализация calc_x2y2z2 для CUDA
__device__ void calc_x2y2z2(int x, int y, int z, int dir, int Lx, int Ly, int Lz, int* x2, int* y2, int* z2) {
    //const int dir_number;
    if (dir < 0 || dir >= dir_number) {
        printf("calc_x2y2z2: invalid dir=%d\n", dir);
        *x2 = x; *y2 = y; *z2 = z;
        return;
    }
    int factor = (z % 2 == 0) ? 1 : -1;
    const int dx[dir_number] = {1, 1, -1, -1, 0, 2};
    const int dy[dir_number] = {1, -1, 1, -1, 2, 0};
    const int dz[dir_number] = {1, -1, -1, 1, 2, 2};
    
    *x2 = x + factor * dx[dir];
    *y2 = y + factor * dy[dir];
    *z2 = z + factor * dz[dir];
    
    if (*x2 < 0) *x2 += Lx;
    if (*x2 >= Lx) *x2 -= Lx;
    if (*y2 < 0) *y2 += Ly;
    if (*y2 >= Ly) *y2 -= Ly;
    *z2 = max(2, min(Lz - 3, *z2));
    printf("calc_x2y2z2: x=%d, y=%d, z=%d, dir=%d, x2=%d, y2=%d, z2=%d\n", x, y, z, dir, *x2, *y2, *z2);
}

// Ядро для обновления смещений атомов
__global__ void axyz_kernel(
    atom_t* atoms,
    int Lx, int Ly, int Lz,
    int* xs, int* ys, int* zs,
    int count,
    hiprandState* states,
    float T,
    float* d_AA_,
    float* d_BB,
    float* d_transform_array,
    int* d_ochered_count,
    int* d_ochered_x,
    int* d_ochered_y,
    int* d_ochered_z,
    int max_ochered_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;

    int x = xs[idx];
    int y = ys[idx];
    int z = zs[idx];
    int atom_idx = z * Lx * Ly + y * Lx + x;

    if (atom_idx >= Lx * Ly * Lz || atoms[atom_idx].type == 0) return;

    unsigned short config_ = atoms[atom_idx].config;
    if (config_ >= Nconfig || config_ == 65535) return;

    float* AA_ptr = d_AA_ + config_ * 6;
    float A_xx = AA_ptr[0];
    float A_yy = AA_ptr[1];
    float A_zz = AA_ptr[2];
    float A_xy = AA_ptr[3];
    float A_xz = AA_ptr[4];
    float A_yz = AA_ptr[5];

    float Bx = atoms[atom_idx].B0.x;
    float By = atoms[atom_idx].B0.y;
    float Bz = atoms[atom_idx].B0.z;

    //const int dir_number;
    for (int dir = 0; dir < dir_number; dir++) {
        int x2, y2, z2;
        calc_x2y2z2(x, y, z, dir, Lx, Ly, Lz, &x2, &y2, &z2);
        int neighbor_idx = z2 * Lx * Ly + y2 * Lx + x2;

        if (neighbor_idx >= 0 && neighbor_idx < Lx * Ly * Lz) {
            float ax2 = atoms[neighbor_idx].a.x;
            float ay2 = atoms[neighbor_idx].a.y;
            float az2 = atoms[neighbor_idx].a.z;

            float* BB_ptr = d_BB + config_ * dir_number * 9 + dir * 9;
            float Bxx = BB_ptr[0]; float Bxy = BB_ptr[1]; float Bxz = BB_ptr[2];
            float Byx = BB_ptr[3]; float Byy = BB_ptr[4]; float Byz = BB_ptr[5];
            float Bzx = BB_ptr[6]; float Bzy = BB_ptr[7]; float Bzz = BB_ptr[8];

            Bx += Bxx * ax2 + Bxy * ay2 + Bxz * az2;
            By += Byx * ax2 + Byy * ay2 + Byz * az2;
            Bz += Bzx * ax2 + Bzy * ay2 + Bzz * az2;

            int ochered_idx = atomicAdd(d_ochered_count, 1);
            if (ochered_idx < max_ochered_size) {
                d_ochered_x[ochered_idx] = x2;
                d_ochered_y[ochered_idx] = y2;
                d_ochered_z[ochered_idx] = z2;
            }
        }
    }

    int ochered_idx = atomicAdd(d_ochered_count, 1);
    if (ochered_idx < max_ochered_size) {
        d_ochered_x[ochered_idx] = x;
        d_ochered_y[ochered_idx] = y;
        d_ochered_z[ochered_idx] = z;
    }

    hiprandState local_state = states[idx];
    float ax_, ay_, az_;
    random_displacements_gpu(&ax_, &ay_, &az_, config_, d_transform_array, &local_state, T);
    states[idx] = local_state;

    ax_ -= 0.5f * (A_xx * Bx + A_xy * By + A_xz * Bz);
    ay_ -= 0.5f * (A_xy * Bx + A_yy * By + A_yz * Bz);
    az_ -= 0.5f * (A_xz * Bx + A_yz * By + A_zz * Bz);

    atoms[atom_idx].a.x = ax_;
    atoms[atom_idx].a.y = ay_;
    atoms[atom_idx].a.z = az_;
}

// Инициализация генераторов случайных чисел
__global__ void setup_kernel(hiprandState* state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

// Внешняя функция для вызова из C++
extern "C" void cuda_do_many_axyz(
    struct coord* atoms_to_update,
    int count,
    atom_t* host_atoms,
    int Lx, int Ly, int Lz,
    float T,
    float* host_AA_,
    float* host_BB,
    float* host_transform_array)
{
    atom_t* dev_atoms;
    int *dev_xs, *dev_ys, *dev_zs;
    hiprandState* dev_states;
    float *dev_AA_, *dev_BB, *dev_transform_array;
    int *dev_ochered_count, *dev_ochered_x, *dev_ochered_y, *dev_ochered_z;

    size_t atoms_size = Lx * Ly * Lz * sizeof(atom_t);
    size_t AA_size = Nconfig * 6 * sizeof(float);
    size_t BB_size = Nconfig * 6 * 9 * sizeof(float);
    size_t transform_size = Nconfig * 6 * sizeof(float);
    int max_ochered_size = count * 7;

    hipMalloc(&dev_atoms, atoms_size);
    hipMalloc(&dev_xs, count * sizeof(int));
    hipMalloc(&dev_ys, count * sizeof(int));
    hipMalloc(&dev_zs, count * sizeof(int));
    hipMalloc(&dev_states, count * sizeof(hiprandState));
    hipMalloc(&dev_AA_, AA_size);
    hipMalloc(&dev_BB, BB_size);
    hipMalloc(&dev_transform_array, transform_size);
    hipMalloc(&dev_ochered_count, sizeof(int));
    hipMalloc(&dev_ochered_x, max_ochered_size * sizeof(int));
    hipMalloc(&dev_ochered_y, max_ochered_size * sizeof(int));
    hipMalloc(&dev_ochered_z, max_ochered_size * sizeof(int));

    hipMemcpy(dev_atoms, host_atoms, atoms_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_AA_, host_AA_, AA_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_BB, host_BB, BB_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_transform_array, host_transform_array, transform_size, hipMemcpyHostToDevice);

    int* host_xs = new int[count];
    int* host_ys = new int[count];
    int* host_zs = new int[count];
    for (int i = 0; i < count; i++) {
        host_xs[i] = atoms_to_update[i].x;
        host_ys[i] = atoms_to_update[i].y;
        host_zs[i] = atoms_to_update[i].z;
    }
    hipMemcpy(dev_xs, host_xs, count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_ys, host_ys, count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_zs, host_zs, count * sizeof(int), hipMemcpyHostToDevice);
    delete[] host_xs;
    delete[] host_ys;
    delete[] host_zs;

    int zero = 0;
    hipMemcpy(dev_ochered_count, &zero, sizeof(int), hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((count + block.x - 1) / block.x);
    setup_kernel<<<grid, block>>>(dev_states, time(NULL));
    hipDeviceSynchronize();

    axyz_kernel<<<grid, block>>>(dev_atoms, Lx, Ly, Lz, dev_xs, dev_ys, dev_zs, count, dev_states, T,
                                 dev_AA_, dev_BB, dev_transform_array,
                                 dev_ochered_count, dev_ochered_x, dev_ochered_y, dev_ochered_z, max_ochered_size);
    hipDeviceSynchronize();

    hipMemcpy(host_atoms, dev_atoms, atoms_size, hipMemcpyDeviceToHost);
    int ochered_count;
    hipMemcpy(&ochered_count, dev_ochered_count, sizeof(int), hipMemcpyDeviceToHost);
    int* host_ochered_x = new int[max_ochered_size];
    int* host_ochered_y = new int[max_ochered_size];
    int* host_ochered_z = new int[max_ochered_size];
    hipMemcpy(host_ochered_x, dev_ochered_x, max_ochered_size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_ochered_y, dev_ochered_y, max_ochered_size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_ochered_z, dev_ochered_z, max_ochered_size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < min(ochered_count, max_ochered_size); i++) {
        v_ochered_Edef(host_ochered_x[i], host_ochered_y[i], host_ochered_z[i]);
    }

    delete[] host_ochered_x;
    delete[] host_ochered_y;
    delete[] host_ochered_z;
    hipFree(dev_atoms);
    hipFree(dev_xs);
    hipFree(dev_ys);
    hipFree(dev_zs);
    hipFree(dev_states);
    hipFree(dev_AA_);
    hipFree(dev_BB);
    hipFree(dev_transform_array);
    hipFree(dev_ochered_count);
    hipFree(dev_ochered_x);
    hipFree(dev_ochered_y);
    hipFree(dev_ochered_z);
}